#include <assert.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>

#include <iostream>

__device__ void convolute(uint16_t* image, uint16_t* kernel, uint16_t* sum_out, uint32_t image_size,
                          uint32_t kernel_size, uint32_t x, uint32_t y, uint32_t padding, uint32_t line_size) {
    *sum_out = 0;
    for (uint32_t desloc_i = 0; desloc_i < kernel_size; desloc_i++) {
        for (uint32_t desloc_j = 0; desloc_j < kernel_size; desloc_j++) {
            uint32_t image_desloc = (x - padding + desloc_i) + ((y - padding + desloc_j) * line_size);
            // printf("%u\n", image_desloc);
            uint16_t pixel = image[image_desloc];
            uint16_t kernel_pixel = kernel[desloc_i + kernel_size * desloc_j];
            if (x == 1 && y == 1) {
                // printf("%lf %lf\n", image[0], kernel[0]);
            }
            *sum_out += pixel * kernel_pixel;
        }
    }
}

__global__ void par_convolution(uint16_t* image, uint16_t* kernel, uint16_t* out, uint32_t image_size,
                                uint32_t kernel_size) {
    uint32_t padding = (kernel_size - 1) / 2;

    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x + padding;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y + padding;

    // printf("%u %u %u\n", x, y, padding);

    uint32_t line_size = blockDim.x * gridDim.x + 2 * padding;

    uint32_t tid = x + (y * line_size);

    uint16_t sum = 0;
    convolute(image, kernel, &sum, image_size, kernel_size, x, y, padding, line_size);
    assert(sum > 0);
    out[tid] = sum / (uint16_t)(kernel_size * kernel_size);
    assert(*(out + tid) == sum / (uint16_t)(kernel_size * kernel_size));
}

void convolution(uint16_t* image, uint32_t image_size, uint16_t* kernel, uint32_t kernel_size, uint16_t* out) {
    uint32_t padding = (kernel_size - 1) / 2;

    // TODO(Otavio): Create a better logic for grid and block dims size
    // Make it in a way that (image_size - 2 * padding) is ways divisible
    // Aka, all convuluted pixels should be processed, no more no less
    dim3 grid(4, 4);
    dim3 block((image_size - 2 * padding) / 4, (image_size - 2 * padding) / 4);

    uint16_t *image_in_device, *image_out_device, *kernel_device;

    hipMalloc(&image_in_device, sizeof(uint16_t) * image_size * image_size);
    // uint16_t* cache = (uint16_t*)malloc(sizeof(uint16_t) * image_size * image_size);
    // for (uint32_t i = 0; i < image_size; i++) {
    //     memcpy(cache + (i * image_size), *(image + i), image_size * sizeof(uint16_t));
    // }
    hipMemcpy(image_in_device, image, image_size * image_size * sizeof(uint16_t), hipMemcpyHostToDevice);

    hipMalloc(&image_out_device, sizeof(uint16_t) * image_size * image_size);
    hipMemcpy(image_out_device, image_in_device, image_size * image_size * sizeof(uint16_t), hipMemcpyDeviceToDevice);
    // for (uint32_t i = 0; i < image_size; i++) {
    //     cudaMemcpy(image_out_device + (i * image_size), *(image + i), image_size * sizeof(uint32_t),
    //                cudaMemcpyHostToDevice);
    // }

    // uint16_t* cache_kernel = (uint16_t*)malloc(sizeof(uint16_t) * kernel_size * kernel_size);
    hipMalloc(&kernel_device, sizeof(uint16_t) * kernel_size * kernel_size);
    // for (uint32_t i = 0; i < kernel_size; i++) {
    //     memcpy(cache_kernel + (i * kernel_size), *(kernel + i), kernel_size * sizeof(uint16_t));
    // }
    // for (size_t i = 0; i < kernel_size; i++) {
    //     for (size_t j = 0; j < kernel_size; j++) {
    //         printf("%lf\n", *(cache_kernel + i + j));
    //     }
    // }
    hipError_t err =
        hipMemcpy(kernel_device, kernel, kernel_size * kernel_size * sizeof(uint16_t), hipMemcpyHostToDevice);

    for (size_t i = 0; i < 10000000; i++) {
        par_convolution<<<grid, block>>>(image_in_device, kernel_device, image_out_device, image_size, kernel_size);
    }
    hipDeviceSynchronize();

    hipMemcpy(out, image_out_device, image_size * image_size * sizeof(uint16_t), hipMemcpyDeviceToHost);

    // for (uint32_t i = 0; i < image_size; i++) {
    //     memcpy(*(out + i), cache + (i * image_size), image_size * sizeof(uint16_t));
    // }
    // free(cache_kernel);
    // free(cache);
    hipDeviceReset();
}

void show_matrix(uint16_t** matrix, uint32_t size) {
    for (uint32_t i = 0; i < size; i++) {
        for (uint32_t j = 0; j < size; j++) {
            printf("%.2u\t|", matrix[i][j]);
        }
        putchar('\n');
    }
}

int main(int argc, char const* argv[]) {
    uint32_t kernel_size = 100;
    uint32_t padding = (kernel_size - 1) / 2;
    uint32_t n = 10000;
    uint32_t image_size = 4 * n + 2 * padding;

    srand(345);
    printf("STARTING CREATION\n");

    uint16_t* image = (uint16_t*)malloc(sizeof(uint16_t) * image_size * image_size);
    printf("END ALLOC\n");
    for (uint32_t i = 0; i < image_size; i++) {
        // image[i] = (uint16_t*)malloc(sizeof(uint16_t) * image_size);
        for (uint32_t j = 0; j < image_size; j++) {
            image[i + j * image_size] = (uint16_t)(i + 1);
        }
    }
    for (uint32_t i = 0; i < image_size; i++) {
        image[i] = 0;
        image[image_size * (image_size - 1) + i] = 0;
        image[image_size * i] = 0;
        image[image_size * i + (image_size - 1)] = 0;
    }
    printf("END CREATION\n");

    uint16_t* image_out = (uint16_t*)malloc(sizeof(uint16_t) * image_size * image_size);
    printf("END ALLOC\n");
    for (uint32_t i = 0; i < image_size * image_size; i++) {
        // image_out[i] = (uint16_t*)malloc(sizeof(uint16_t) * image_size);
        // for (uint32_t j = 0; j < image_size; j++) {
        image_out[i] = 0;
        // }
    }

    uint16_t* kernel = (uint16_t*)malloc(sizeof(uint16_t*) * kernel_size * kernel_size);
    for (uint32_t i = 0; i < kernel_size; i++) {
        // kernel[i] = (uint16_t*)malloc(sizeof(uint16_t) * kernel_size);
        for (uint32_t j = 0; j < kernel_size; j++) {
            kernel[i + j * kernel_size] = 1;
        }
    }
    // kernel[padding][padding] = 1;

    // show_matrix(image, image_size);
    // show_matrix(kernel, kernel_size);
    printf("STARTING CONV\n");
    convolution(image, image_size, kernel, kernel_size, image_out);
    printf("END CONV\n");
    // printf("\n");
    // show_matrix(image, image_size);
    // printf("\n");
    // show_matrix(image_out, image_size);
    // printf("%lf", *(*(image_out + 10) + 100));

    // for (uint32_t i = 0; i < image_size; i++) {
    //     free(image[i]);
    // }
    free(image);

    // for (uint32_t i = 0; i < image_size; i++) {
    //     free(image_out[i]);
    // }
    free(image_out);

    // for (uint32_t i = 0; i < kernel_size; i++) {
    //     free(kernel[i]);
    // }
    free(kernel);

    return 0;
}
