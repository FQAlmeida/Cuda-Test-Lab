#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <math.h>
#include <stdio.h>

#include <iostream>

#include "../alloc_image_mem/alloc_image_mem.hpp"
#include "convolution.cuh"
#include "cuda_common.cuh"

__device__ void convolute(float* image, float* kernel, float* sum_out, uint32_t image_size, uint32_t kernel_size,
                          uint32_t x, uint32_t y, uint32_t padding, uint32_t line_size) {
    *sum_out = 0;
    for (uint32_t desloc_i = 0; desloc_i < kernel_size; desloc_i++) {
        for (uint32_t desloc_j = 0; desloc_j < kernel_size; desloc_j++) {
            uint32_t image_desloc = (x - padding + desloc_i) + ((y - padding + desloc_j) * line_size);
            float pixel = image[image_desloc];
            float kernel_pixel = kernel[desloc_i + kernel_size * desloc_j];
            *sum_out += pixel * kernel_pixel;
        }
    }
}

__global__ void par_convolution(float* image, float* kernel, float* out, uint32_t image_size, uint32_t kernel_size) {
    uint32_t padding = (kernel_size - 1) / 2;

    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x + padding;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y + padding;

    uint32_t line_size = blockDim.x * gridDim.x + 2 * padding;

    uint32_t tid = x + (y * line_size);

    // printf("%d %d %d\n", tid, x, y);

    float sum = 0;
    convolute(image, kernel, &sum, image_size, kernel_size, x, y, padding, line_size);
    assert(sum > 0);
    // printf("%f\n", sum);
    out[tid] = sum / (float)(kernel_size * kernel_size);
}

__global__ void checker(float* image, float* image_out, uint32_t image_size) {
    for (size_t i = 1; i < image_size - 1; i++) {
        for (size_t j = 1; j < image_size - 1; j++) {
            assert(image[i + j * image_size] != image_out[i + j * image_size]);
            // printf("%f %f\n", image[i + j * image_size], image_out[i + j * image_size]);
        }
    }
}

void convolution(float* image, uint32_t image_size, float* kernel, uint32_t kernel_size, float* out) {
    uint32_t padding = (kernel_size - 1) / 2;
    // TODO(Otavio): Create a better logic for grid and block dims size
    // Make it in a way that (image_size - 2 * padding) is ways divisible
    // Aka, all convuluted pixels should be processed, no more no less
    dim3 grid(32, 32);
    dim3 block((image_size - 2 * padding) / 32, (image_size - 2 * padding) / 32);
    printf("Thread Block Size: %d %d\n", (image_size - 2 * padding) / 32, (image_size - 2 * padding) / 32);

    float *image_in_device, *image_out_device, *kernel_device;

    gpuErrchk(hipMalloc(&image_in_device, sizeof(float) * image_size * image_size));
    gpuErrchk(hipMemcpy(image_in_device, image, image_size * image_size * sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&image_out_device, sizeof(float) * image_size * image_size));
    // hipMemcpy(image_out_device, image_in_device, image_size * image_size * sizeof(float),
    // hipMemcpyDeviceToDevice);
    gpuErrchk(hipMalloc(&kernel_device, sizeof(float) * kernel_size * kernel_size));
    gpuErrchk(hipMemcpy(kernel_device, kernel, kernel_size * kernel_size * sizeof(float), hipMemcpyHostToDevice));
    printf("STARTING CONV LOOP\n");

    // for (size_t i = 0; i < 10000; i++) {
    //     for (size_t j = 0; j < 1000; j++) {
    par_convolution<<<grid, block>>>(image_in_device, kernel_device, image_out_device, image_size, kernel_size);
    gpuErrchk(hipGetLastError());
    //     }
    // }

    gpuErrchk(hipDeviceSynchronize());
    printf("END CONV LOOP\n");

    checker<<<1, 1>>>(image_in_device, image_out_device, image_size);
    gpuErrchk(hipGetLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(out, image_out_device, image_size * image_size * sizeof(float), hipMemcpyDeviceToHost));

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipDeviceReset());
}

void show_matrix(float* matrix, uint32_t size, uint32_t max_print) {
    for (uint32_t i = 0; i < min(size, max_print); i++) {
        for (uint32_t j = 0; j < min(size, max_print); j++) {
            printf("%.2f\t|", matrix[i + j * size]);
        }
        putchar('\n');
    }
}

void save_matrix(float* matrix, uint32_t size, uint32_t max_print, const char* filename) {
    FILE* fptr;
    fptr = fopen(filename, "w+");
    if (fptr == NULL) {
        printf("Error while opening file %s!\n", filename);
        exit(1);
    }
    for (uint32_t i = 0; i < min(size, max_print); i++) {
        for (uint32_t j = 0; j < min(size, max_print); j++) {
            fprintf(fptr, "%.2f\t|", matrix[i + j * size]);
        }
        fputc('\n', fptr);
    }
    fclose(fptr);
}

int run_convolution() {
    uint32_t kernel_size = 3;
    uint32_t padding = (kernel_size - 1) / 2;
    uint32_t n = 32;
    uint32_t image_size = 32 * n + 2 * padding;

    srand(345);
    printf("STARTING CREATION\n");

    float* image = alloc_image(image_size);
    float* image_out = alloc_image_out(image_size);
    float* kernel = alloc_kernel(kernel_size);

    save_matrix(image, image_size, image_size, "data/convolution_matrix.txt");
    printf("STARTING CONV\n");
    convolution(image, image_size, kernel, kernel_size, image_out);
    printf("END CONV\n");

    show_matrix(image, image_size, 15);
    printf("----------------------------------------------\n");
    show_matrix(image_out, image_size, 15);

    save_matrix(image_out, image_size, image_size, "data/convolution_matrix_out.txt");

    free(image);
    free(image_out);
    free(kernel);

    return 0;
}
