#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>

#include <iostream>

__global__ void par_convolution(uint32_t* image, uint32_t* kernel,
                                uint32_t** out, uint32_t image_size,
                                uint32_t kernel_size) {
    // uint32_t tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y) +
    //    (gridDim.x * blockDim.x * threadIdx.y) +
    //    (blockDim.x * blockIdx.x) + threadIdx.x;

    uint32_t padding = (kernel_size - 1) / 2;

    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x + padding;
    uint32_t y = threadIdx.y + blockDim.y * blockIdx.y + padding;

    uint32_t line_size = blockDim.x * gridDim.x;
    uint32_t col_size = blockDim.y * gridDim.y;

    uint32_t tid = x + (y * line_size);

    // if (x <= 0 || x >= line_size - 1) {
    //     return;
    // } else if (y <= 0 || y >= col_size - 1) {
    //     return;
    // }

    printf("%d %u %u %u %u (%u %u)\n", tid, threadIdx.x, threadIdx.y,
           blockIdx.x, blockIdx.y, x, y);
}

void convolution(uint8_t** image, uint32_t image_size, uint8_t** kernel,
                 uint32_t kernel_size, uint8_t*** out) {
    dim3 grid(4, 4);
    uint32_t padding = (kernel_size - 1) / 2;
    dim3 block((image_size - 2 * padding) / 4, (image_size - 2 * padding) / 4);

    uint32_t *image_in_device, *image_out_device, *kernel_device;

    hipMalloc((void**)&image_in_device,
               sizeof(uint8_t) * image_size * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy(image_in_device + (i * image_size), image + i, image_size,
                   hipMemcpyHostToDevice);
    }

    hipMalloc(&image_out_device, sizeof(uint8_t*) * image_size * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy(image_out_device + (i * image_size), image + i, image_size,
                   hipMemcpyHostToDevice);
    }

    hipMalloc(&kernel_device, sizeof(uint8_t*) * kernel_size * kernel_size);
    for (uint32_t i = 0; i < kernel_size; i++) {
        hipMemcpy(kernel_device + (i * kernel_size), kernel + i, kernel_size,
                   hipMemcpyHostToDevice);
    }

    par_convolution<<<grid, block>>>(image_in_device, kernel_device,
                                     &image_out_device, image_size,
                                     kernel_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy((*out) + i, image_out_device + (i * image_size), image_size,
                   hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipDeviceReset();
}

void show_matrix(uint8_t** matrix, uint32_t size) {
    for (uint32_t i = 0; i < size; i++) {
        for (uint32_t j = 0; j < size; j++) {
            printf("%u\t|", matrix[i][j]);
        }
        putchar('\n');
    }
}

int main(int argc, char const* argv[]) {
    uint32_t kernel_size = 3;
    uint32_t padding = (kernel_size - 1) / 2;
    uint32_t n = 4;
    uint32_t image_size = 4 * (n) + 2 * padding;

    uint8_t** image = (uint8_t**)malloc(sizeof(uint8_t*) * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        image[i] = (uint8_t*)malloc(sizeof(uint8_t) * image_size);
        for (uint32_t j = 0; j < image_size; j++) {
            image[i][j] = 100;
        }
    }
    for (uint32_t i = 0; i < image_size; i++) {
        image[i][0] = 0;
        image[i][image_size - 1] = 0;
        image[0][i] = 0;
        image[image_size - 1][i] = 0;
    }

    uint8_t** image_out = (uint8_t**)malloc(sizeof(uint8_t*) * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        image_out[i] = (uint8_t*)malloc(sizeof(uint8_t) * image_size);
        for (uint32_t j = 0; j < image_size; j++) {
            image_out[i][j] = 0;
        }
    }

    uint8_t** kernel = (uint8_t**)malloc(sizeof(uint8_t*) * kernel_size);
    for (uint32_t i = 0; i < kernel_size; i++) {
        kernel[i] = (uint8_t*)malloc(sizeof(uint8_t) * kernel_size);
        for (uint32_t j = 0; j < kernel_size; j++) {
            kernel[i][j] = 0;
        }
    }
    kernel[padding][padding] = 1;

    // show_matrix(image, image_size);
    // show_matrix(kernel, kernel_size);

    convolution(image, image_size, kernel, kernel_size, &image_out);

    // show_matrix(image_out, image_size);

    for (uint32_t i = 0; i < image_size; i++) {
        free(image[i]);
    }
    free(image);

    for (uint32_t i = 0; i < image_size; i++) {
        free(image_out[i]);
    }
    free(image_out);

    for (uint32_t i = 0; i < kernel_size; i++) {
        free(kernel[i]);
    }
    free(kernel);

    return 0;
}
