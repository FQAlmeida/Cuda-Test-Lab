#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>

__global__ void par_convolution(uint32_t* image, uint32_t* kernel,
                                uint32_t** out, uint32_t image_size,
                                uint32_t kernel_size) {
    uint32_t tid = (blockDim.x * blockDim.y * gridDim.x * blockIdx.y) +
                   (gridDim.x * blockDim.x * threadIdx.y) +
                   (blockDim.x * blockIdx.x) + threadIdx.x;
    printf("%d %u %u %u %u\n", tid, threadIdx.x, threadIdx.y, blockIdx.x,
           blockIdx.y);
}

void convolution(uint8_t** image, uint32_t image_size, uint8_t** kernel,
                 uint32_t kernel_size, uint8_t*** out) {
    dim3 grid(4, 4);
    dim3 block(image_size / 4, image_size / 4);

    uint32_t *image_in_device, *image_out_device, *kernel_device;

    hipMalloc((void**)&image_in_device,
               sizeof(uint8_t) * image_size * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy(image_in_device + (i * image_size), image + i, image_size,
                   hipMemcpyHostToDevice);
    }

    hipMalloc(&image_out_device, sizeof(uint8_t*) * image_size * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy(image_out_device + (i * image_size), image + i, image_size,
                   hipMemcpyHostToDevice);
    }

    hipMalloc(&kernel_device, sizeof(uint8_t*) * kernel_size * kernel_size);
    for (uint32_t i = 0; i < kernel_size; i++) {
        hipMemcpy(kernel_device + (i * kernel_size), kernel + i, kernel_size,
                   hipMemcpyHostToDevice);
    }

    par_convolution<<<grid, block>>>(image_in_device, kernel_device,
                                     &image_out_device, image_size,
                                     kernel_size);
    for (uint32_t i = 0; i < image_size; i++) {
        hipMemcpy((*out) + i, image_out_device + (i * image_size), image_size,
                   hipMemcpyDeviceToHost);
    }

    hipDeviceSynchronize();
    hipDeviceReset();
}

void show_matrix(uint8_t** matrix, uint32_t size) {
    for (uint32_t i = 0; i < size; i++) {
        for (uint32_t j = 0; j < size; j++) {
            printf("%u\t|", matrix[i][j]);
        }
        putchar('\n');
    }
}

int main(int argc, char const* argv[]) {
    uint32_t image_size = 16;
    uint8_t** image = (uint8_t**)malloc(sizeof(uint8_t*) * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        image[i] = (uint8_t*)malloc(sizeof(uint8_t) * image_size);
        for (uint32_t j = 0; j < image_size; j++) {
            image[i][j] = 0;
        }
    }

    uint8_t** image_out = (uint8_t**)malloc(sizeof(uint8_t*) * image_size);
    for (uint32_t i = 0; i < image_size; i++) {
        image_out[i] = (uint8_t*)malloc(sizeof(uint8_t) * image_size);
        for (uint32_t j = 0; j < image_size; j++) {
            image_out[i][j] = 0;
        }
    }

    uint32_t kernel_size = 3;
    uint8_t** kernel = (uint8_t**)malloc(sizeof(uint8_t*) * kernel_size);
    for (uint32_t i = 0; i < kernel_size; i++) {
        kernel[i] = (uint8_t*)malloc(sizeof(uint8_t) * kernel_size);
        for (uint32_t j = 0; j < kernel_size; j++) {
            kernel[i][j] = 1;
        }
    }

    show_matrix(image, image_size);
    show_matrix(kernel, kernel_size);

    convolution(image, image_size, kernel, kernel_size, &image_out);

    show_matrix(image_out, image_size);

    for (uint32_t i = 0; i < image_size; i++) {
        free(image[i]);
    }
    free(image);

    for (uint32_t i = 0; i < image_size; i++) {
        free(image_out[i]);
    }
    free(image_out);

    for (uint32_t i = 0; i < kernel_size; i++) {
        free(kernel[i]);
    }
    free(kernel);

    return 0;
}
